
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void ArraySum(int* dev_arr, int* result) {
    int id = threadIdx.x;
    atomicAdd(result, dev_arr[id]);
}

#define N 100
int main(int argc, char* argv[]) {
    int arr[N];
    for (int i = 0; i < N; i++) {
        arr[i] = i + 1;
        printf("%d ", arr[i]);
    }
    printf("\n");

    int result = 0;
    int *dev_arr, *dev_result;
    hipMalloc(&dev_arr, sizeof(arr));
    hipMalloc(&dev_result, sizeof(int));
    hipMemcpy(dev_arr, arr, sizeof(arr), hipMemcpyHostToDevice);
    ArraySum<<<1, N>>>(dev_arr, dev_result);
    hipMemcpy(&result, dev_result, sizeof(result), hipMemcpyDeviceToHost);
    printf("Sum: %d\n", result);
    return 0;
}

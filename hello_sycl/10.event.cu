
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void kernel_dummy_1() { printf("kernel_dummy_1\n"); }
__global__ void kernel_dummy_2() { printf("kernel_dummy_2\n"); }
__global__ void kernel_dummy_3() { printf("kernel_dummy_3\n"); }
__global__ void kernel_dummy_4() { printf("kernel_dummy_4\n"); }

int main(int argc, char *argv[]) {
    hipStream_t stream_1;
    hipStream_t stream_2;
    hipStreamCreate(&stream_1);
    hipStreamCreate(&stream_2);

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);

    kernel_dummy_1<<<1, 1, 0, stream_1>>>();

    hipEventRecord(start_event, stream_1);
    kernel_dummy_2<<<1, 1, 0, stream_1>>>();
    hipEventRecord(stop_event, stream_1);

    kernel_dummy_3<<<1, 1, 0, stream_2>>>();

    float runtime = 0.0;
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&runtime, start_event, stop_event);
    printf("runtime %f\n", runtime);

    kernel_dummy_4<<<1, 1, 0, stream_2>>>();

    hipDeviceSynchronize();
    return 0;
}
